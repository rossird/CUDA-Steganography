#include <fstream>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <math.h>

using namespace std;

//Execute 1 thread per pixel of output image.
//Requires no atomics
__global__ void encode_per_pixel_kernel(uchar4* const d_destImg,
                              const char* const d_binData,
                              int numBytesData)
{
  int pixel = threadIdx.x + blockDim.x * blockIdx.x;
  if(pixel >= numBytesData)
    return;
  
  //Pixel 5 is at byte 3.
  int dataStart = pixel / 2 + 1;
  int nibble = pixel % 2;

  char dataByte = d_binData[dataStart];
  
  //Can't do next part in a loop because we have to access differently (x,y,z,w)
  
  //Channel 0 (first bit in the nibble)
  int offset = (7 - 1 * nibble);
  char mask = 1 << offset;
  char bit = (dataByte & mask) >> offset;
  d_destImg[pixel].x += bit;
  
  //Channel 1 (2nd bit)
  offset -= 1;
  mask >>= 1;
  bit = (dataByte & mask) >> offset;
  d_destImg[pixel].y += bit;
  
  //Channel 2 (3rd bit)
  offset -= 1;
  mask >>= 1;
  bit = (dataByte & mask) >> offset;
  d_destImg[pixel].z += bit;
  
  //Channel 3 (4th bit)
  offset -= 1;
  mask >>= 1;
  bit = (dataByte & mask) >> offset;
  d_destImg[pixel].z += bit;
  
}


//1 channel per bit of data
//8 channels per byte of data
__global__ void encode_per_channel_kernel(uchar4* const d_destImg,
                              const char* const d_binData,
                              int numBytesData)
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if(idx >= 8 * numBytesData)
    return;
    
  //4 channels
  int channel = idx % 4;
  int pixel = idx / 4;
  
  //Pixel 5 is at byte 3.
  int dataStart = pixel / 2 + 1;
  int nibble = pixel % 2;
  
  char dataByte = d_binData[dataStart];
  
  //Get the bit
  int offset = (7 - 1 * nibble);
  char mask = 1 << offset;
  char bit = (dataByte & mask) >> offset;
  
  if(channel == 0) {
    d_destImg[pixel].x += bit;
  } else if(channel == 1){ 
    d_destImg[pixel].y += bit;
  } else if(channel == 2){
    d_destImg[pixel].z += bit;
  } else if(channel == 3){
    d_destImg[pixel].w += bit;
  }
 
}

/**

| 10 11 12 15 ; 11 255 12 0 |
| 15 10 13 5  ; 15 14 19 80 | Original image (each set of 4 is 1 pixel).
| 12 14 16 21 ; 14 18 10 16 |
| 10 10 10 10 ; 10 10 10 10 |

+

[ 1001 0110 1111 0000 1010 0101 0100 1100]  Data file

= 

| 11 11 12 16 ; 11 0  13 0  |
| 15 11 14 6  ; 15 14 19 80 | Encoded image
| 13 14 16 21 ; 14 19 10 17 |
| 10 11 10 10 ; 11 11 10 10 |
 
 */
void encode_parallel(const uchar4* const h_sourceImg,
                     uchar4* const h_destImg,
                     const char* const h_binData,
                     int numBytesData,
                     const size_t numRowsSource, const size_t numColsSource)
{

  //Allocate device memory
  uchar4* d_destImg;
  char* d_binData;
  hipMalloc(&d_destImg, sizeof(uchar4) * numRowsSource * numColsSource);
  hipMalloc(&d_binData, sizeof(char) * numBytesData);
  
  hipMemcpy(d_destImg, h_sourceImg, sizeof(uchar4) * numRowsSource * numColsSource, hipMemcpyHostToDevice); 
  hipMemcpy(d_binData, h_binData, numBytesData, hipMemcpyHostToDevice);

  //Each thread handles 1 pixel
  //This means 1 thread per 4 bits of data (2 threads per byte)
  int numThreads = numBytesData * 2.0;
  int threadsPerBlock = 1024;
  int numBlocks = ceil((float)numThreads / threadsPerBlock);
  //cout << "numBlocks: " << numBlocks << " blockSize: " << blockSize << " numThreads: " << numThreads << endl;
  
  encode_per_pixel_kernel<<<numBlocks, threadsPerBlock>>>(d_destImg, d_binData, numBytesData);
  
  
  //Each thread handles 1 channel of 1 pixel
  //This means 1 thread per bit of data (8 threads per byte)
  numThreads = numBytesData * 8;
  threadsPerBlock = 1024;
  numBlocks = ceil((float)numThreads / threadsPerBlock);
  
  cout << "numBlocks: " << numBlocks << " blockSize: " << threadsPerBlock << " numThreads: " << numThreads << endl;
  
  //encode_per_channel_kernel<<<numBlocks, threadsPerBlock>>>(d_destImg, d_binData, numBytesData);
  
  hipMemcpy(h_destImg, d_destImg, sizeof(uchar4) * numRowsSource * numColsSource, hipMemcpyDeviceToHost);
  
  //Free memory
  hipFree(d_destImg);
  hipFree(d_binData);
                  
}
