#include <fstream>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <math.h>

using namespace std;

/**
| 11 11 12 16 ; 11 0  13 0  |
| 15 11 14 6  ; 15 14 19 80 | Encoded image (each set of 4 is 1 pixel)
| 13 14 16 21 ; 14 19 10 17 |
| 10 11 10 10 ; 11 11 10 10 |

-

| 10 11 12 15 ; 11 255 12 0 |
| 15 10 13 5  ; 15 14 19 80 | Original image 
| 12 14 16 21 ; 14 18 10 16 |
| 10 10 10 10 ; 10 10 10 10 |

=

[ 1001 0110 1111 0000 1010 0101 0100 1100]  Data file


 */
void decode_parallel(const uchar4* const h_encodedImage,
                     const uchar4* const h_originalImage,
                     char* const h_binData,
                     int numBytesData,
                     const size_t numRowsSource, const size_t numColsSource)
{

                  
}
